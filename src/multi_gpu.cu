#include "hip/hip_runtime.h"
#include <mpi.h>

#include "macro.h"
#include "multi_gpu.h"

#define MAX_PROCESS_NUM 16

#define print(condition, format, ...) \
  do                                  \
    if (condition) {                  \
      printf(format, ##__VA_ARGS__);  \
    }                                 \
  while (0)

static int mpi_rank;
static int mpi_size;

static int gpu_num;
static int my_gpu_id;

constexpr int ARRAY_SIZE = 1024 * 1024;
static int *local_memory;
static int *neighbor_memory;
static int *redundant_memory;

__global__ void init_arry(int *arr, int proc_rank, int length) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = idx; i < length; i += stride) {
    arr[i] = i * 100 + proc_rank;
  }
}

__global__ void print_array(int *arr, int proc_rank, int length) {
  // int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = blockDim.x * gridDim.x;
  //    for (int i = idx; i < length; i += stride) {
  //      printf("%d ", arr[i]);
  //    }
  if (length == 0) {
    printf(
        "in function print_arr, my mpi rank is %d, first four elems are \
        %d %d %d %d\nlast four elems are %d %d %d %d\n",
        proc_rank, arr[0], arr[1], arr[2], arr[3], arr[ARRAY_SIZE - 4], arr[ARRAY_SIZE - 3], arr[ARRAY_SIZE - 2],
        arr[ARRAY_SIZE - 1]);
  }
}

void multi_gpu_test() {
  int src_rank;
  int dst_rank;

  MPI_Request send_req;
  MPI_Request recv_req;

  CHECK_MPI(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  CHECK_MPI(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));
  CHECK_CUDA(hipGetDeviceCount(&gpu_num));
  src_rank = (mpi_rank + mpi_size - 1) % mpi_size;
  dst_rank = (mpi_rank + 1) % mpi_size;
  my_gpu_id = mpi_rank % gpu_num;
  CHECK_CUDA(hipSetDevice(my_gpu_id));

  if (mpi_size == gpu_num) {
    print(mpi_rank == 0, "mpi_size and gpu_num is the same %d\n", mpi_size);
  } else {
    printf("mpi_size = %d, mpi_rank = %d, gpu_id = %d\n", mpi_size, mpi_rank, my_gpu_id);
  }

  // init local memory
  CHECK_CUDA(hipMalloc(&local_memory, ARRAY_SIZE * sizeof(int)));
  CHECK_CUDA(hipMalloc(&redundant_memory, ARRAY_SIZE * sizeof(int)));
  init_arry<<<ARRAY_SIZE / 256, 256>>>(local_memory, mpi_rank, ARRAY_SIZE);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
  print(mpi_rank == 0, "data init over\n");

  // comm
  // 创建显存句柄，并发送或接收
  hipIpcMemHandle_t handle;
  hipIpcMemHandle_t pre_handle;
  hipIpcGetMemHandle(&handle, local_memory);

  // int MPI_Isend(const void *buf, int count, MPI_Datatype datatype, int dest,
  // int tag, MPI_Comm comm, MPI_Request *request)
  MPI_Isend(&handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, dst_rank, 0, MPI_COMM_WORLD, &send_req);
  // int MPI_Irecv(void *buf, int count, MPI_Datatype datatype, int source, int
  // tag, MPI_Comm comm, MPI_Request *request)
  MPI_Irecv(&pre_handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, src_rank, 0, MPI_COMM_WORLD, &recv_req);
  MPI_Wait(&send_req, MPI_STATUS_IGNORE);
  MPI_Wait(&recv_req, MPI_STATUS_IGNORE);
  CHECK_CUDA(hipIpcOpenMemHandle((void **)&neighbor_memory, pre_handle, hipIpcMemLazyEnablePeerAccess));

  CHECK_CUDA(hipMemcpy(redundant_memory, neighbor_memory, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToDevice));
  // print
  print_array<<<1, 1>>>(redundant_memory, mpi_rank, 0);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  CHECK_CUDA(hipFree(local_memory));
  CHECK_CUDA(hipFree(redundant_memory));
  CHECK_CUDA(hipIpcCloseMemHandle(neighbor_memory));
}